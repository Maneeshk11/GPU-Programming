
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel() {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("%d.\n",i * i);
}

int main(int argc, char** argv) {
        int numBl = atoi(argv[1]);
        int numTh = atoi(argv[2]);
        dkernel<<<numBl, numTh>>>();
        hipDeviceSynchronize();
        return 0;
}

