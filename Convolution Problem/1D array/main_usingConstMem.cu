#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

int *Arr, *ker, *ans;
int *d_Arr, *d_ker, *d_ans;
int masked_Device;

__global__ void findUsingGPUs(int *d_arr, int *d_ker, int n, int m, int *d_ans) {	
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
    // int masked_Device = m/2;
    int k = blockDim.x * gridDim.x;
    int i = tid;
    int start = i - masked_Device;
    while (i < n) {
        for (int j=0;j<m;j++) {
            if (start >= 0 && start<n) {
                d_ans[i] += d_arr[start]*d_ker[j];

            }
            start++;
        }
        i+=k;
        start = i - maskedLength;
    }        
}

int main(int argc, char* argv[]) {
    // srand(0);

    int n = atoi(argv[1]);
    int m = atoi(argv[2]);
    int blocks = atoi(argv[3]);
    int threads = atoi(argv[4]);
    int number = atoi(argv[5]);

    Arr = (int*)calloc(n, sizeof(int));
    ans = (int*)calloc(n, sizeof(int));
    ker = (int*)calloc(m, sizeof(int));
    // ans = (int*)malloc(100, sizeof(int));

    for (int i = 0; i < n; i++) {
        Arr[i] = 1;
        ans[i] = 0;
    }
    for (int i = 0; i < m; i++) {
        ker[i] = 0;
    }
    int last = m-1;
    while (number > 0) {
        ker[i] = number%2;
        last--;
        number/=2;
    }

    hipMalloc((void **)&d_Arr, sizeof(int)*n);
    hipMalloc((void **)&d_ans, sizeof(int)*n);
	hipMalloc((void **)&d_ker, sizeof(int)*m);

    hipMemcpy(d_Arr, Arr, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(d_ans, ans, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(d_ker, ker, sizeof(int)*m, hipMemcpyHostToDevice);

    // int total = blocks * threads;
    int maskedLength = m/2;
    hipMemcpyToSymbol(HIP_SYMBOL(masked_Device), &maskedLength, sizeof(maskedLength));


    findUsingGPUs<<<blocks, threads>>>(d_Arr, d_ker, n, m, d_ans);
    hipDeviceSynchronize();

    hipMemcpy(ans, d_ans, sizeof(int)*n, hipMemcpyDeviceToHost);

    // for (int i = 0; i < n; i++) {
    //     printf("%d ", ans[i]);
    // }
}