#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

int **Arr, **ker, **ans;
int **d_Arr, **d_ker, **d_ans;

__global__ void findUsingGPUs(int **d_arr, int **d_ker, int n, int m, int **d_ans) {	
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int maskedLength = m/2; // mid point of the kernel array
    int k = blockDim.x * gridDim.x;
    int i = tid;
    int start = i - maskedLength;
    while (i < n) {
        for (int j=0;j<m;j++) {
            for (int k=0;k<m;k++) {
                if (start >= 0 && start<n) {
                    d_ans[i] += d_arr[start]*d_ker[j];

                }
            }
            start++;
        }
        i+=k;
        start = i - maskedLength;
    }        
}

int main(int argc, char* argv[]) {
    // srand(0);

    int n = atoi(argv[1]);
    int m = atoi(argv[2]);
    int blocks = atoi(argv[3]);
    int threads = atoi(argv[4]);

    Arr = (int **)calloc(sizeof(int *), n);
    ans = (int **)calloc(sizeof(int *), n);
    ker = (int **)calloc(sizeof(int *), m);

    for(int i=0;i<n;i++) {
		Arr[i]=(int *)calloc(sizeof(int), n);
		ans[i]=(int *)calloc(sizeof(int), n);
	}
    for (int i=0;i<m;i++)  {
        ker[i] = (int *)calloc(sizeof(int), m);
    }

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            Arr[i][j] = 1;
            ans[i][j] = 0;
        }
    }

    for (int i = 0; i < m; i++) {
        for (int j=0;j<m;j++) {
            ker[i][j] = 1;
        }
    }

    hipMalloc((void***)&d_Arr,  n*sizeof(int *));
    hipMalloc((void***)&d_ans,  n*sizeof(int *));
    hipMalloc((void***)&d_ker,  m*sizeof(int *));

    for (int i = 0; i < n; i++) {
        hipMalloc((void**) &(d_Arr[i]), n*sizeof(int));
        hipMemcpy (d_Arr[i], Arr[i], n*sizeof(int), hipMemcpyHostToDevice);
        hipMalloc((void**) &(d_ans[i]), n*sizeof(int));
        hipMemcpy (d_ans[i], ans[i], n*sizeof(int), hipMemcpyHostToDevice);
    }
    for (int i=0;i<m;i++) {
        hipMalloc((void**) &(d_ker[i]), m*sizeof(int));
        hipMemcpy (d_ker[i], ker[i], m*sizeof(int), hipMemcpyHostToDevice);
    }

    // int total = blocks * threads;
    // int maskedLength = m/2;

    findUsingGPUs<<<blocks, threads>>>(d_Arr, d_ker, n, m, d_ans);
    hipDeviceSynchronize();

    for (int i=0;i<n;i++) {
        hipMemcpy(ans[i], d_ans[i], sizeof(int)*n, hipMemcpyDeviceToHost);
    }

    

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++){
            printf("%d ", ans[i][j]);
        }
        printf("\n");
    }
}