#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

int* A, *arr, *ans;
int *DA, *Darr;

_global_ void findUsingGPUs(int *DA, int *Darr, int n, int total) {	
	 int tid = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = tid; i < n; i += total) {
        atomicAdd(&Darr[DA[i]], 1);
    }
}

int main(int argc, char* argv[]) {
    srand(0);

    int size=atoi(argv[1]);
    int blocks = atoi(argv[2]);
    int threads = atoi(argv[3]);
    int n = size;

    A = (int*)calloc(n, sizeof(int));
    arr = (int*)calloc(100, sizeof(int));
    // ans = (int*)malloc(100, sizeof(int));

    for (int i = 0; i < n; i++) {
        A[i] = rand()%100;
    }
    for (int i = 0; i < 100; i++) {
        arr[i] = 0;
    }
    hipMalloc((void **)&DA, sizeof(int)*n);
	hipMalloc((void **)&Darr, sizeof(int)*100);
    hipMemcpy(DA, A, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(Darr, arr, sizeof(int)*100, hipMemcpyHostToDevice);
    int total = blocks * threads;

    findUsingGPUs<<<blocks, threads>>>(DA, Darr, n, total);
    hipDeviceSynchronize();
    hipMemcpy(arr, Darr, sizeof(int)*100, hipMemcpyDeviceToHost);


    int comp = 0;
    for (int i = 0; i < 100; i++) {
        printf("%d marks : \n", arr[i]);
        comp += arr[i];
    }
    printf("\n\n\nTEST(total no of students) = %d", comp);
}