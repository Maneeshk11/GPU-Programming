#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel(int* darr) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	darr[i] = i * i;        
	//printf("%d.\n",i * i);
}

int main(int argc, char** argv) {
	int numBl = atoi(argv[1]);
	int numTh = atoi(argv[2]);
	int *arr, *darr;
	int N = numBl * numTh;
	arr = (int*)calloc(N, sizeof(int));
	hipMalloc(&darr, sizeof(int) * N);
        dkernel<<<numBl, numTh>>>(darr);
	hipDeviceSynchronize();
	hipMemcpy(arr, darr, N*sizeof(int), hipMemcpyDeviceToHost);
	
	for (int i=0; i<N;i++) {
		printf("%d\n", arr[i]);
	}	
	return 0;
}
