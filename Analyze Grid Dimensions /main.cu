#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel() {
    printf("Block: (%d, %d, %d)\tThread: (%d, %d, %d)\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
}

int main(int argc, char** argv) {
    dim3 grid(2, 2, 2);
    dim3 block(2, 2, 2);

    dkernel<<<grid, block>>>();
    hipDeviceSynchronize();
}